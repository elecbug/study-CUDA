
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <thread>
#include <chrono>

using namespace std;
using namespace std::chrono;

void task1(string msg)
{
    cout << "task1 says: " << msg;
}

void use_thread()
{
    thread t1(task1, "Hello");
    t1.join();
}

void xor_cpu(size_t *dst, size_t *src, size_t* size)
{
    for (int i = 0; i < *size; i++)
    {
        dst[i] = dst[i] ^ src[i];
    }
}

__global__
void xor_gpu(size_t *dst, size_t *src, size_t* size)
{
    for (int i = 0; i < *size; i++)
    {
        dst[i] = dst[i] ^ src[i];
    }
}

void check_time_cpu(int tried)
{
    milliseconds ms1 = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    size_t size = 1 << 20;
    size_t *src, *dst;
        
    src = (size_t*)malloc(sizeof(size_t) * size);
    dst = (size_t*)malloc(sizeof(size_t) * size);

    for (int i = 0; i < tried; i++)
    {
        xor_cpu(dst, src, &size);
    }

    free(src);
    free(dst);

    milliseconds ms2 = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    printf("CPU: %lld\n", (ms2 - ms1).count());
}

void check_time_gpu(int tried)
{
    milliseconds ms1 = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    size_t size = 1 << 20;
    size_t *src, *dst;
        
    hipMalloc(&src, sizeof(size_t) * size);
    hipMalloc(&dst, sizeof(size_t) * size);

    for (int i = 0; i < tried; i++)
    {
        xor_gpu<<<2048, 32>>>(dst, src, &size);
    }

    hipFree(src);
    hipFree(dst);

    milliseconds ms2 = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    printf("GPU: %lld\n", (ms2 - ms1).count());
}

int main()
{
    check_time_cpu(1000);
    check_time_gpu(1000);
}
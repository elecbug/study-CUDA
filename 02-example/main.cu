
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add(int *a, int *b, int* result)
{
    *result = *a + *b;
}

int main(void)
{
    int *x, *y, result;
    int *d_x, *d_y, *d_result;

    x = (int*)malloc(sizeof(int));
    y = (int*)malloc(sizeof(int));

    hipMalloc(&d_x, sizeof(int));
    hipMalloc(&d_y, sizeof(int));
    hipMalloc(&d_result, sizeof(int));
    
    *x = 1;
    *y = 2;

    hipMemcpy(d_x, x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(int), hipMemcpyHostToDevice);

    add<<<1024, 16>>>(d_x, d_y, d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", result);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    free(x);
    free(y);
}